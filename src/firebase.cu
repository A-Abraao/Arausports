#include "hip/hip_runtime.h"
import { initializeApp } from "firebase/app";
import { doc, setDoc, getDoc, updateDoc, increment, runTransaction } from "firebase/firestore";
import { getStorage, ref as storageRef, uploadBytes, getDownloadURL } from "firebase/storage";
import { 
  getAuth, 
  GoogleAuthProvider, 
  onAuthStateChanged, 
  type User, 
  createUserWithEmailAndPassword,
  sendEmailVerification,
  linkWithCredential,
  EmailAuthProvider,
  fetchSignInMethodsForEmail,
  signInWithEmailAndPassword,
  signOut
} from "firebase/auth";
import { updateProfile } from "firebase/auth";
import { getFirestore } from "firebase/firestore";

export const uploadProfileImage = async (uid: string, file: File) => {
  const ref = storageRef(storage, `profile_pictures/${uid}/${file.name}`);
  await uploadBytes(ref, file);
  const url = await getDownloadURL(ref);
  await updateDoc(doc(db, "users", uid), { photoURL: url });
  return url;
};

const firebaseConfig = {
  apiKey: import.meta.env.VITE_FIREBASE_API_KEY,
  authDomain: import.meta.env.VITE_FIREBASE_AUTH_DOMAIN,
  projectId: import.meta.env.VITE_FIREBASE_PROJECT_ID,
  storageBucket: import.meta.env.VITE_FIREBASE_STORAGE_BUCKET,
  messagingSenderId: import.meta.env.VITE_FIREBASE_MESSAGING_SENDER_ID,
  appId: import.meta.env.VITE_FIREBASE_APP_ID,
};

const app = initializeApp(firebaseConfig);

export const storage = getStorage(app);

export const auth = getAuth(app);
export const db = getFirestore(app);

export const googleProvider = new GoogleAuthProvider();

export const onAuth = (cb: (user: User | null) => void) => onAuthStateChanged(auth, cb);


export const updateUserBio = async (uid: string, bio: string) => {
    const userRef = doc(db, "users", uid);
    
    const snap = await getDoc(userRef);
    if (!snap.exists()) {
      await setDoc(userRef, {
        uid,
        bio,
        createdAt: new Date().toISOString(),
      });
    } else {
      await updateDoc(userRef, {
        bio,
        updatedAt: new Date().toISOString(),
      });
    }
};

export const updateUserProfile = async (
  uid: string,
  data: { bio?: string; username?: string }
) => {
  const userRef = doc(db, "users", uid);
  
  const snap = await getDoc(userRef);

  const dataToUpdateInFirestore: { bio?: string; displayName?: string; updatedAt: string } = {
    updatedAt: new Date().toISOString()
  };

  if (data.bio) {
    dataToUpdateInFirestore.bio = data.bio;
  }
  if (data.username) {
    dataToUpdateInFirestore.displayName = data.username;
  }

  if (snap.exists()) {
    await updateDoc(userRef, dataToUpdateInFirestore);
  } else {
    
    await setDoc(userRef, {
      uid,
      ...dataToUpdateInFirestore,
      createdAt: new Date().toISOString(),
    });
  }

  if (data.username) {
    const user = auth.currentUser;
    if (user) {
      await updateProfile(user, { displayName: data.username });
    }
  }
};

const getUserData = async (uid: string) => {
  const userRef = doc(db, "users", uid);
  const snap = await getDoc(userRef);
  return snap.exists() ? snap.data() : null;
};

export const signUpWithEmail = async (username: string, email: string, password: string) => {
  const methods = await fetchSignInMethodsForEmail(auth, email);

  if (methods.length > 0) {
    if (methods.includes("password")) {
      throw new Error("E-mail já cadastrado com e-mail/senha. Faça login ou recupere a senha.");
    }

    if (methods.includes("google.com")) {
      const current = auth.currentUser;

      if (current && current.email?.toLowerCase() === email.toLowerCase()) {
        const credential = EmailAuthProvider.credential(email, password);
        try {
          const linkedUserCred = await linkWithCredential(current as User, credential);

          if (!linkedUserCred.user.emailVerified) {
            await sendEmailVerification(linkedUserCred.user);
          }
          return linkedUserCred.user;
        } catch (err: any) {
          throw new Error(err?.message ?? "Erro ao vincular credencial.");
        }
      } else {
        throw new Error(
          "Já existe uma conta com esse e-mail usando Google. Faça login com Google para vincular uma senha."
        );
      }
    }

    throw new Error(
      `E-mail já cadastrado com outro provedor: ${methods.join(", ")}. Utilize esse provedor para entrar.`
    );
  }

  const userCredential = await createUserWithEmailAndPassword(auth, email, password);

  if (userCredential.user) {
    await updateProfile(userCredential.user, { displayName: username });

    await sendEmailVerification(userCredential.user);
  }

  return userCredential.user;
};

export const createUserDocIfNotExists = async (
  user: User,
  extraData?: { username?: string; photoURL?: string }
) => {
  const userRef = doc(db, "users", user.uid);
  const snap = await getDoc(userRef);

  
  const displayName = extraData?.username ?? user.displayName ?? null;
  const photoURL = extraData?.photoURL ?? user.photoURL ?? null;

  if (!snap.exists()) {
    await setDoc(userRef, {
      uid: user.uid,
      email: user.email ?? null,
      displayName,
      username: extraData?.username ?? null,
      photoURL,
      createdAt: new Date().toISOString(),
    });
  } else {
    const existing = snap.data() as any;
    const updates: any = {};
    if (displayName && !existing.displayName) updates.displayName = displayName;
    if (photoURL && !existing.photoURL) updates.photoURL = photoURL;
    if (Object.keys(updates).length) {
      await updateDoc(userRef, updates);
    }
  }

  if (displayName && user.displayName !== displayName) {
    try {
      await updateProfile(user, { displayName });
    } catch (err) {
    
    }
  }
};

export const signInWithEmail = async (email: string, password: string) => {
  const userCredential = await signInWithEmailAndPassword(auth, email, password);

  if (!userCredential.user.emailVerified) {
    await signOut(auth); 
    throw new Error("Verifique seu e-mail antes de acessar.");
  }

  return userCredential.user;
};

export const resendVerification = async (user: User) => {
  if (user && !user.emailVerified) {
    await sendEmailVerification(user);
  }
};

export const entrarNoEvento = async (
  eventoId: string,
  ownerId: string,
  participantId: string
) => {
  const eventoRef = doc(db, "users", ownerId, "eventos", eventoId);
  const participanteRef = doc(db, "users", ownerId, "eventos", eventoId, "participantes", participantId);

  await runTransaction(db, async (tx) => {
    const eventoSnap = await tx.get(eventoRef);
    if (!eventoSnap.exists()) {
      throw new Error("Evento não existe.");
    }

    const eventoData = eventoSnap.data() as any;
    const capacidade = Number(eventoData.capacidade ?? 0);
    const participantesAtuais = Number(eventoData.participantesAtuais ?? 0);

    if (capacidade <= 0) {
      throw new Error("Capacidade do evento inválida.");
    }

    if (participantesAtuais >= capacidade) {
      throw new Error("Evento lotado.");
    }

    const participanteSnap = await tx.get(participanteRef);
    if (participanteSnap.exists()) {
      throw new Error("Você já está inscrito nesse evento.");
    }

    tx.update(eventoRef, { participantesAtuais: increment(1) });
    tx.set(participanteRef, {
      userId: participantId,
      joinedAt: new Date().toISOString(),
    });
  });
};

export const sairDoEvento = async (
  eventoId: string,
  ownerId: string,
  participantId: string
) => {
  const eventoRef = doc(db, "users", ownerId, "eventos", eventoId);
  const participanteRef = doc(db, "users", ownerId, "eventos", eventoId, "participantes", participantId);

  await runTransaction(db, async (tx) => {
    const participanteSnap = await tx.get(participanteRef);
    if (!participanteSnap.exists()) {
      throw new Error("Você não está inscrito nesse evento.");
    }

    const eventoSnap = await tx.get(eventoRef);
    if (!eventoSnap.exists()) {
      throw new Error("Evento não existe.");
    }

    const eventoData = eventoSnap.data() as any;
    const participantesAtuais = Number(eventoData.participantesAtuais ?? 0);

    const novoValor = Math.max(0, participantesAtuais - 1);

    tx.update(eventoRef, { participantesAtuais: novoValor });
    tx.delete(participanteRef);
  });
};